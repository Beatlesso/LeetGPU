#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

const int TileSize = 32;
const int multiB = 8;

// A(M, N) B(N, K) C(M, K)
__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    __shared__ float TileA[TileSize][TileSize];
    __shared__ float TileB[TileSize][TileSize * multiB];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float sum[multiB];
    for(int i = 0 ; i < multiB ; ++ i) sum[i] = 0;
    // 当前 C 中 TileBlock 的左上角 行列坐标
    int row = blockDim.y * blockIdx.y;
    int col = blockDim.x * blockIdx.x * multiB;
    for(int i = 0 ; i < (N + TileSize - 1) / TileSize ; ++ i) { // 枚举 TileA TileB
        int Arow = row + ty;
        int Acol = i * TileSize + tx;
        int Brow = i * TileSize + ty;
        int Bcol = col + tx;
        int Aptr = Arow * N + Acol;
        int Bptr = Brow * K + Bcol;

        if(Arow < M && Acol < N) TileA[ty][tx] = A[Aptr];
        else TileA[ty][tx] = 0;
        if(Brow < N) {
            for(int j = 0 ; j < multiB ; ++ j) {
                if (Bcol + TileSize * j < K) TileB[ty][tx + TileSize * j] = *(B + Bptr + TileSize * j);
                else TileB[ty][tx + TileSize * j] = 0;
            }
        }

        __syncthreads();
        for(int j = 0 ; j < TileSize ; ++ j) {
            for(int k = 0 ; k < multiB ; ++ k) {
                sum[k] += TileA[ty][j] * TileB[j][tx + TileSize * k];
            }
        }
        __syncthreads();
    }
    int Crow = row + ty, Ccol = col + tx;
    int Cptr = Crow * K + Ccol;
    if(Crow < M) {
        for(int i = 0 ; i < multiB ; ++ i) {
            if(Ccol + TileSize * i < K) {
                C[Cptr + TileSize * i] = sum[i];
            }
        }
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(TileSize, TileSize);
    dim3 blocksPerGrid((K + (TileSize * multiB) - 1) / (TileSize * multiB),
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
