#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

const int BLOCK_SIZE = 16;
// 非常 naive 的版本, 访存不连续
__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if(x < rows && y < cols) {
        output[y * rows + x] = input[x * cols + y];
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int rows, int cols) {
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((rows + BLOCK_SIZE - 1) / BLOCK_SIZE,
                       (cols + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}